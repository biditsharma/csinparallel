#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

/* kernel function */
__global__ void MatrixKernel(float *dM, float *dN, float *dP, int width) {

    /* calculate the row index of the dP element and M */
    // TO DO
    // int row = .........
    // end TO DO

    /* calculate the column index of dP element and N */
    // TO DO
    // int col = .........
    // end TO DO

    float pvalue = 0.0f;
    for (int k = 0; k < width; k++) {
        float M_elem = dM[row * width + k];
        float N_elem = dN[k * width + col];
        pvalue += M_elem * N_elem;
    }
    dP[row * width + col] = pvalue;
}

/* function that you will call in mpi code */
extern "C" void MatrixMul(float* M, float* N, float* P, int width, int block_size) {

    int matrix_size = width * width * sizeof(float);
    float *dM, *dN, *dP;

    // Allocate and Load M and N to device memory
    hipMalloc(&dM, matrix_size);
    hipMemcpy(dM, M, matrix_size, hipMemcpyHostToDevice);

    hipMalloc(&dN, matrix_size);
    hipMemcpy(dN, N, matrix_size, hipMemcpyHostToDevice);

    // Allocate P on device
    hipMalloc(&dP, matrix_size);

    dim3 dimGrid(width/block_size, width/block_size);
    dim3 dimBlock(block_size, block_size);

    // TO DO
    // call the kernel function
    // end TO DO

    hipMemcpy(P, dP, matrix_size, hipMemcpyDeviceToHost);

    hipFree(dP);
    hipFree(dM);
    hipFree(dN);
}
