#include <stdio.h>
#include <hip/hip_runtime.h>

/* kernel function */
__global__ void kernel(int *a, int *b, int *c, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size)
        c[index] = a[index] + b[index];
}

/* function to be called in the MPI program, and size is the number of elements in array */
extern "C" void run_kernel(int *a, int *b, int *c, int size, int nblocks, int nthreads) {

    /* pointers to the arrays on the GPU */
    int *dev_a, *dev_b, *dev_c; 

    /* Allocate memory on the GPU */
    hipMalloc((void**)&dev_a, sizeof(int)*size);
    hipMalloc((void**)&dev_b, sizeof(int)*size);
    hipMalloc((void**)&dev_c, sizeof(int)*size);

    /* Copy array a and b from host to GPU */
    hipMemcpy(dev_a, a, sizeof(int)*size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, sizeof(int)*size, hipMemcpyHostToDevice);

    /* Calling the kernel function to do calculation */
    kernel<<<nblocks, nthreads>>>(dev_a, dev_b, dev_c, size);

    /* Copy the result array from device to host*/
    hipMemcpy(c, dev_c, sizeof(int)*size, hipMemcpyDeviceToHost);

    /* Free memory on the device */
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
