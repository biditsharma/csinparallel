#include "hip/hip_runtime.h"
/* Parallelization: Infectious Disease
 * By Yu Zhao, Macalester College
 * July 2013 */

#ifndef PANDEMIC_CUDA_CU
#define PANDEMIC_CUDA_CU

#include "Defaults.h"

#include <hiprand.h>     // cuda random number gen lib
#include <time.h>       // seed the random number generator 

/********************* global variable *********************/
// variables needed for cuda random number generator
hiprandGenerator_t gen;      // cuda random number generator
time_t current_time;        // time needed as seed
float *rand_nums;           // array pointer for rand number
/***********************************************************/

/* CUDA shared memory allocation */
extern __shared__ int array[];

/*
    move()
        Spawns threads to move everyone randomly
*/
__global__ void cuda_move(char *states_dev, int *x_locations_dev, int *y_locations_dev, 
    char DEAD, int environment_width, int environment_height, float *rand_nums, int SIZE)
{
    // set up thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // If the person is not dead, then
    if(states_dev[id] != DEAD){

        // The thread randomly picks whether the person moves left
        // or right or does not move in the x dimension
        int x_move_direction = (int)(rand_nums[id]*3) - 1;

        // The thread randomly picks whether the person moves up 
        // or down or does not move in the y dimension
        int y_move_direction = (int)(rand_nums[id+SIZE]*3) - 1;

        // If the person will remain in the bounds of the
        // environment after moving, then
        if( (x_locations_dev[id] + x_move_direction >= 0) && 
            (x_locations_dev[id] + x_move_direction < environment_width) && 
            (y_locations_dev[id] + y_move_direction >= 0) &&
            (y_locations_dev[id] + y_move_direction < environment_height) )
        {
            // The thread moves the person
            x_locations_dev[id] = x_locations_dev[id] + x_move_direction;
            y_locations_dev[id] = y_locations_dev[id] + y_move_direction;
        }
    }
}

/*
    cuda_susceptible()
        Spawns threads to handle those that are ssusceptible by 
        deciding whether or not they should be marked infected.
*/
__global__ void cuda_susceptible(char *states_dev, int *x_locations_dev, 
    int *y_locations_dev, int *infected_x_locations_dev, 
    int *infected_y_locations_dev, int *num_infected_dev, 
    int *num_susceptible_dev, int *num_infection_attempts_dev,
    int *num_infections_dev, float *rand_nums, int global_num_infected, 
    int infection_radius, int contagiousness_factor, char SUSCEPTIBLE, char INFECTED)
{
    // set up thread id, block id and block dimension
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int blockId = threadIdx.x;
    int numThread = blockDim.x;

    // counters
    int i, num_infected_nearby;

    // set up shared memory
    int *num_infected = (int*)array; 
    int *num_susceptible = (int*)&num_infected[numThread];
    #ifdef SHOW_RESULTS
    int *num_infection_attempts = (int*)&num_susceptible[numThread];
    int *num_infections = (int*)&num_infection_attempts[numThread];
    #endif

    // reset the shared memory
    num_infected[blockId] = 0;
    num_susceptible[blockId] = 0;
    #ifdef SHOW_RESULTS
    num_infection_attempts[blockId] = 0;
    num_infections[blockId] = 0;
    #endif

    // If the person is susceptible, then
    if(states_dev[id] == SUSCEPTIBLE)
    {
        // For each of the infected people (received earlier from 
        // all processes) or until the number of infected people 
        // nearby is 1, the thread does the following
        num_infected_nearby = 0;
        for(i=0; i<=global_num_infected-1 && num_infected_nearby<1; i++)
        {
            // If this person is within the infection radius, then
            if( (x_locations_dev[id] > infected_x_locations_dev[i] - infection_radius) && 
                (x_locations_dev[id] < infected_x_locations_dev[i] + infection_radius) && 
                (y_locations_dev[id] > infected_y_locations_dev[i] - infection_radius) &&
                (y_locations_dev[id] < infected_y_locations_dev[i] + infection_radius) )
            {
                // The thread increments the number of infected people nearby
                num_infected_nearby++;
            }
        }

        if(num_infected_nearby >= 1){
            #ifdef SHOW_RESULTS
            num_infection_attempts[blockId]++;
            #endif
        }
        
        // generate a random number between 0 and 100
        int rand_num = (int)(rand_nums[id]*100);

        // If there is at least one infected person nearby, and 
        // a random number less than 100 is less than or equal 
        // to the contagiousness factor, then
        if(num_infected_nearby >= 1 && rand_num <= contagiousness_factor)
        {
            // The thread changes person1’s state to infected
            states_dev[id] = INFECTED;
            // The thread updates the counters
            num_infected[blockId]++;
            num_susceptible[blockId]--;
            #ifdef SHOW_RESULTS
            num_infections[blockId]++;
            #endif
        }
    }

    __syncthreads();
    // if we have numThread to the power of 2, we can use binary
    // tree reduction to increase performance
    if(((numThread!=0) && !(numThread & (numThread-1)))){
        i = numThread/2;
        while (i != 0) {
            if (blockId < i){
                num_infected[blockId] += num_infected[blockId + i];
                num_susceptible[blockId] += num_susceptible[blockId + i];
                #ifdef SHOW_RESULTS
                num_infection_attempts[blockId] += num_infection_attempts[blockId + i];
                num_infections[blockId] += num_infections[blockId + i];
                #endif
            }
            __syncthreads();
            i /= 2; 
        }
    }
    // Else, we can only add-up results in shared memory using
    // the first thread in each block
    else{
        if(blockId == 0) {
            for(i=1; i<numThread; i++){
                num_infected[0] += num_infected[i];
                num_susceptible[0] += num_susceptible[i];
                #ifdef SHOW_RESULTS
                num_infection_attempts[0] += num_infection_attempts[i];
                num_infections[0] += num_infections[i];
                #endif
            }
        }
    }

    // use atomicAdd function to add the results to device pointers
    if(blockId == 0) {
        atomicAdd(num_infected_dev, num_infected[0]);
        atomicAdd(num_susceptible_dev, num_susceptible[0]);
        #ifdef SHOW_RESULTS
        atomicAdd(num_infection_attempts_dev, num_infection_attempts[0]);
        atomicAdd(num_infections_dev, num_infections[0]);
        #endif
    }
}

/*
    cuda_infected()
        Spawns threads to handle infected personales
*/
__global__ void cuda_infected(char *states_dev, int *num_days_infected_dev, 
    int *num_recovery_attempts_dev, int *num_deaths_dev, 
    int *num_infected_dev, int *num_immune_dev, int *num_dead_dev,
    int duration_of_disease, int deadliness_factor, char IMMUNE, char DEAD, 
    char INFECTED, float *rand_nums)
{
    // set up thread id, block id and block dimension
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int blockId = threadIdx.x;
    int numThread = blockDim.x;

    // counter
    int i;

    // set up shared memory
    int* num_infected = (int*)array; 
    int* num_immune = (int*)&num_infected[numThread];
    int* num_dead = (int*)&num_immune[numThread];
    #ifdef SHOW_RESULTS
    int* num_recovery_attempts = (int*)&num_dead[numThread];
    int* num_deaths = (int*)&num_recovery_attempts[numThread];
    #endif

    // reset the shared memory
    num_infected[blockId] = 0;
    num_immune[blockId] = 0;
    num_dead[blockId] = 0;
    #ifdef SHOW_RESULTS
    num_recovery_attempts[blockId] = 0;
    num_deaths[blockId] = 0;
    #endif

    // If the person is infected and has been for the full 
    // duration of the disease, then
    if(states_dev[id] == INFECTED && num_days_infected_dev[id] == duration_of_disease)
    {
        #ifdef SHOW_RESULTS
        num_recovery_attempts[blockId]++;
        #endif

        // generate a random number between 0 and 100
        int rand_num = (int)(rand_nums[id]*100);

        // If a random number less than 100 is less than 
        // the deadliness factor, then
        if(rand_num <= deadliness_factor)
        {
            // The thread changes the person’s state to dead 
            states_dev[id] = DEAD;
            // The thread updates the counters
            num_dead[blockId]++;
            num_infected[blockId]--;
            #ifdef SHOW_RESULTS
            num_deaths[blockId]++;
            #endif
        }
        else
        {
            // The thread changes the person’s state to immune
            states_dev[id] = IMMUNE;
            // The thread updates the counters
            num_immune[blockId]++;
            num_infected[blockId]--;
        }
    }

    __syncthreads();

    // if we have numThread to the power of 2, we can use binary
    // tree reduction to increase performance
    if(((numThread!=0) && !(numThread & (numThread-1)))){
        i = numThread/2;
        while (i != 0) {
            if (blockId < i){
                num_infected[blockId] += num_infected[blockId + i];
                num_immune[blockId] += num_immune[blockId + i];
                num_dead[blockId] += num_dead[blockId + i];
                #ifdef SHOW_RESULTS
                num_recovery_attempts[blockId] += num_recovery_attempts[blockId + i];
                num_deaths[blockId] += num_deaths[blockId + i];
                #endif
            }
            __syncthreads();
            i /= 2; 
        }
    }
    // Else, we can only add-up results in shared memory using
    // the first thread in each block
    else{
        if(blockId == 0) {
            for(i=1; i<numThread; i++){
                num_infected[0] += num_infected[i];
                num_immune[0] += num_immune[i];
                num_dead[0] += num_dead[i];
                #ifdef SHOW_RESULTS
                num_recovery_attempts[0] += num_recovery_attempts[i];
                num_deaths[0] += num_deaths[i];
                #endif
            }
        }
    }

    // use atomicAdd function to add the results to device pointers
    if(blockId == 0) {
        atomicAdd(num_infected_dev, num_infected[0]);
        atomicAdd(num_immune_dev, num_immune[0]);
        atomicAdd(num_dead_dev, num_dead[0]);
        #ifdef SHOW_RESULTS
        atomicAdd(num_recovery_attempts_dev, num_recovery_attempts[0]);
        atomicAdd(num_deaths_dev, num_deaths[0]);
        #endif
    }
}

/*
    cuda_update_days_infected()
        Spawns threads to increase infected days
*/
__global__ void cuda_update_days_infected(char *states_dev, int *num_days_infected_dev,
    char INFECTED)
{
    // set up thread id
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // If the person is infected, then
    if(states_dev[id] == INFECTED)
    {
        // Increment the number of days the person has been infected
        num_days_infected_dev[id]++;
    }
}

/*
    cuda_init()
        initialize cuda environment
*/
extern "C" void cuda_init(struct global_t *global, struct our_t *our, struct cuda_t *cuda)
{
    // initialize size needed for cudamalloc operations
    cuda->our_size = sizeof(int) * our->our_number_of_people;
    cuda->their_size = sizeof(int) * global->total_number_of_people;
    cuda->our_states_size = sizeof(char) * our->our_number_of_people;

    // allocate the memory on the GPU
    // arrays in global and our struct
    hipMalloc((void**)&cuda->their_infected_x_locations_dev, cuda->their_size);
    hipMalloc((void**)&cuda->their_infected_y_locations_dev, cuda->their_size);
    hipMalloc((void**)&cuda->our_x_locations_dev, cuda->our_size);
    hipMalloc((void**)&cuda->our_y_locations_dev, cuda->our_size);
    hipMalloc((void**)&cuda->our_states_dev, cuda->our_states_size);
    hipMalloc((void**)&cuda->our_num_days_infected_dev, cuda->our_size);
    // states counters in our struct
    hipMalloc((void**)&cuda->our_num_susceptible_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_immune_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_dead_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_infected_dev, sizeof(int));
    #ifdef SHOW_RESULTS
    // stats variables in stats struct
    hipMalloc((void**)&cuda->our_num_infections_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_infection_attempts_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_deaths_dev, sizeof(int));
    hipMalloc((void**)&cuda->our_num_recovery_attempts_dev, sizeof(int));
    #endif

    // create cuda random number generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    // get time 
    time(&current_time);
    // generate seed for the rand number generator
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long)current_time);
    // array to hold random number
    hipMalloc((void**)&rand_nums, 2 * our->our_number_of_people * sizeof(float));
    
    // set up 1D array for cuda kernel
    // if we have less than 256 people, initialize only people size of threads
    cuda->numThread = (our->our_number_of_people < 256 ? our->our_number_of_people : 256);
    cuda->numBlock = (our->our_number_of_people+cuda->numThread-1)/cuda->numThread;
};

/*
    cuda_run()
        run cuda environment
*/
extern "C" void cuda_run(struct global_t *global, struct our_t *our, 
    struct const_t *constant, struct stats_t *stats, struct cuda_t *cuda)
{
    // copy infected locations to device in EVERY ITERATION
    hipMemcpy(cuda->their_infected_x_locations_dev, global->their_infected_x_locations, cuda->their_size, hipMemcpyHostToDevice);
    hipMemcpy(cuda->their_infected_y_locations_dev, global->their_infected_y_locations, cuda->their_size, hipMemcpyHostToDevice);
    
    // copy other information to device only in FIRST ITERATION
    // we don't need to copy these information every iteration 
    // becuase they can be reused in each iteration without any
    // process at the host end.
    if(our->current_day == 0){
        // copy arrays in our struct
        hipMemcpy(cuda->our_x_locations_dev, our->our_x_locations, cuda->our_size, hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_y_locations_dev, our->our_y_locations, cuda->our_size, hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_states_dev, our->our_states, cuda->our_states_size, hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_days_infected_dev, our->our_num_days_infected, cuda->our_size, hipMemcpyHostToDevice);
        // copy states counters in our struct
        hipMemcpy(cuda->our_num_susceptible_dev, &our->our_num_susceptible, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_immune_dev, &our->our_num_immune, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_dead_dev, &our->our_num_dead, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_infected_dev, &our->our_num_infected, sizeof(int), hipMemcpyHostToDevice);
        
        #ifdef SHOW_RESULTS
        // variables in stats data are initialized as doubles, yet CUDA
        // atomic operations prefer integer than doubles. Therefore, we
        // cast doubles to integer before the hipMemcpy operations.
        cuda->our_num_infections_int = (int)stats->our_num_infections;
        cuda->our_num_infection_attempts_int = (int)stats->our_num_infection_attempts;
        cuda->our_num_deaths_int = (int)stats->our_num_deaths;
        cuda->our_num_recovery_attempts_int = (int)stats->our_num_recovery_attempts;
        // copy stats variables in stats struct
        hipMemcpy(cuda->our_num_infections_dev, &cuda->our_num_infections_int, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_infection_attempts_dev, &cuda->our_num_infection_attempts_int, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_deaths_dev, &cuda->our_num_deaths_int, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(cuda->our_num_recovery_attempts_dev, &cuda->our_num_recovery_attempts_int, sizeof(int), hipMemcpyHostToDevice);
        #endif
    }

    // generate 2 * our_number_of_people many of randome numbers.
    // we need twice many of people number because movement are in
    // both X direction and Y direction
    hiprandGenerateUniform(gen, rand_nums, 2 * our->our_number_of_people);

    // execute device code on updating people's movement
    int environment_width = constant->environment_width;
    int environment_height = constant->environment_height;
    cuda_move<<<cuda->numBlock, cuda->numThread>>>(cuda->our_states_dev, 
        cuda->our_x_locations_dev, cuda->our_y_locations_dev, DEAD, 
        environment_width, environment_height, rand_nums, our->our_number_of_people);
    // Sync Threads
    hipDeviceSynchronize();

    // generate our_number_of_people many of randome numbers.
    hiprandGenerateUniform(gen, rand_nums, our->our_number_of_people);

    // execute device code on susceptible people
    int infection_radius = constant->infection_radius;
    int contagiousness_factor = constant->contagiousness_factor;
    int total_num_infected = global->total_num_infected;
    cuda_susceptible<<<cuda->numBlock, cuda->numThread, 4*cuda->numThread*sizeof(int)>>>(
        cuda->our_states_dev, cuda->our_x_locations_dev, cuda->our_y_locations_dev, 
        cuda->their_infected_x_locations_dev, cuda->their_infected_y_locations_dev, 
        cuda->our_num_infected_dev, cuda->our_num_susceptible_dev, 
        cuda->our_num_infection_attempts_dev, cuda->our_num_infections_dev, 
        rand_nums, total_num_infected, infection_radius, 
        contagiousness_factor, SUSCEPTIBLE, INFECTED);
    // Sync Threads
    hipDeviceSynchronize();

    // generate our_number_of_people many of randome numbers.
    hiprandGenerateUniform(gen, rand_nums, our->our_number_of_people);

    // execute device code on infected people
    int duration_of_disease = constant->duration_of_disease;
    int deadliness_factor = constant->deadliness_factor;
    cuda_infected<<<cuda->numBlock, cuda->numThread, 5*cuda->numThread*sizeof(int)>>>(
        cuda->our_states_dev, cuda->our_num_days_infected_dev, 
        cuda->our_num_recovery_attempts_dev, cuda->our_num_deaths_dev, 
        cuda->our_num_infected_dev, cuda->our_num_immune_dev, 
        cuda->our_num_dead_dev, duration_of_disease, deadliness_factor, 
        IMMUNE, DEAD, INFECTED, rand_nums);
    // Sync Threads
    hipDeviceSynchronize();

    // execute device code to update infected days
    cuda_update_days_infected<<<cuda->numBlock, cuda->numThread>>>(
        cuda->our_states_dev, cuda->our_num_days_infected_dev, INFECTED);
    // Sync Threads
    hipDeviceSynchronize();

    // copy our locations, our states and our_num_infected back to host
    // in EVERY ITERATION
    hipMemcpy(our->our_x_locations, cuda->our_x_locations_dev, cuda->our_size, hipMemcpyDeviceToHost);
    hipMemcpy(our->our_y_locations, cuda->our_y_locations_dev, cuda->our_size, hipMemcpyDeviceToHost);
    hipMemcpy(our->our_states, cuda->our_states_dev, cuda->our_states_size, hipMemcpyDeviceToHost);
    hipMemcpy(&our->our_num_infected, cuda->our_num_infected_dev, sizeof(int), hipMemcpyDeviceToHost);

    // copy other information back to host only in LAST ITERATION
    // we only copy the counters back for results calculation.
    // we don't need to copy our_num_days_infected back.
    if(our->current_day == constant->total_number_of_days){
        // copy states counters in our struct
        hipMemcpy(&our->our_num_susceptible, cuda->our_num_susceptible_dev, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&our->our_num_immune, cuda->our_num_immune_dev, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&our->our_num_dead, cuda->our_num_dead_dev, sizeof(int), hipMemcpyDeviceToHost);
        
        #ifdef SHOW_RESULTS
        // copy stats variables in stats struct
        hipMemcpy(&cuda->our_num_infections_int, cuda->our_num_infections_dev, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&cuda->our_num_infection_attempts_int, cuda->our_num_infection_attempts_dev, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&cuda->our_num_deaths_int, cuda->our_num_deaths_dev, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&cuda->our_num_recovery_attempts_int, cuda->our_num_recovery_attempts_dev, sizeof(int), hipMemcpyDeviceToHost);
        // cast interger back to double after the hipMemcpy operations.
        stats->our_num_infections = (double)cuda->our_num_infections_int;
        stats->our_num_infection_attempts = (double)cuda->our_num_infection_attempts_int;
        stats->our_num_deaths = (double)cuda->our_num_deaths_int;
        stats->our_num_recovery_attempts = (double)cuda->our_num_recovery_attempts_int;
        #endif
    }
}

/*
    cuda_finish()
        clean up cuda environment
*/
extern "C" void cuda_finish(struct cuda_t *cuda)
{
    // free the memory on the GPU
    // arrays in global and our struct
    hipFree(cuda->their_infected_x_locations_dev);
    hipFree(cuda->their_infected_y_locations_dev);
    hipFree(cuda->our_x_locations_dev);
    hipFree(cuda->our_y_locations_dev);
    hipFree(cuda->our_states_dev);
    hipFree(cuda->our_num_days_infected_dev);
    // states counters in our struct
    hipFree(cuda->our_num_susceptible_dev);
    hipFree(cuda->our_num_immune_dev);
    hipFree(cuda->our_num_dead_dev);
    hipFree(cuda->our_num_infected_dev);

    #ifdef SHOW_RESULTS
    // stats variables in stats struct
    hipFree(cuda->our_num_infections_dev);
    hipFree(cuda->our_num_infection_attempts_dev);
    hipFree(cuda->our_num_deaths_dev);
    hipFree(cuda->our_num_recovery_attempts_dev);
    #endif

    // array to hold random number
    hipFree(rand_nums);
    // destroy cuda random number generator
    hiprandDestroyGenerator(gen);
};

#endif